#include <hip/hip_runtime.h>

__global__ void reverse_array(float* input, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N / 2) {
        float temp = input[i];
        input[i] = input[N - 1 - i];
        input[N - 1 - i] = temp;
    }
}

// input is device pointer
extern "C" void solve(float* input, int N) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    reverse_array<<<blocksPerGrid, threadsPerBlock>>>(input, N);
    hipDeviceSynchronize();
}