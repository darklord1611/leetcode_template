#include <hip/hip_runtime.h>

__global__ void leaky_relu_kernel(const float* input, float* output, int N, float alpha) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {
        output[i] = (input[i] > 0) ? input[i] : (input[i] * alpha);
    }
}

// input, output are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(const float* input, float* output, int N) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    float alpha = 0.01;
    
    leaky_relu_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, N, alpha);
    hipDeviceSynchronize();
}